/*
Copyright (c) 2025 Andrés Morán (andres.moran.l@uc.cl)
Licensed under the terms of the MIT License (see ./LICENSE).
*/

#include <hip/hip_runtime.h>

#include <cstdlib>
#include <iostream>
#include <vector>
#include <algorithm> // tests

using std::cout;
using std::vector;
using std::generate;

// Constants

const int DEFAULT_N_THREADS_PER_DIM = 32; // max 1024 per block
const int DEFAULT_SHARED_MEM = 32*32 * 4 * 2; // 32768 bytes

// Structures

struct __align__(16) GaussianEliminationCtx {
  int prime_number;
  int mod_p_pivot_seek_from_row;
  int mod_p_curr_col;
  int mod_p_row_to_push;
  int mod_p_pivot_val; // TODO: possibly unused
};

// Kernels and devices 

/*
  Mod p linear algebra routines for small primes (<MAX_INT)
  TODO:
  - Separate routines for p >> 1, small primes and p = 2
  - Sparse matrix Gaussian elimination (TODO), multiplication (SELLP)
*/

__device__ int positive_modulo(int i, int n) { 
  /*
    Input: i arbitrary, n: modulus, n > 0
    Output: positive i % n representative
  */
  if (n == 2) {
    return i & 0x00000001;
  }
  return (i % n + n) % n;
}

__device__ int mod_2_inverse(int a) {
  return ((a & 0x00000001) << 1) - 1; // Output: -1 if error
}

__device__ int mod_3_inverse(int a) {
  int b = a % 3;
  if (b == 0) return -1;
  return b;
}

__device__ int mod_p_inverse(int p, int a) {
  /*
    Extended Euclidean division
    Mod p multiplicative inverse
    Output: x_1 = a^{-1}
  */
  int u = a;
  int v = p;
  
  if (p == 2)
    return mod_2_inverse(a);

  if (p == 3)
    return mod_3_inverse(a);

  if (u % v == 0) {
    return -1;
  }

  u = positive_modulo(a, p);
  
  int x_1 = 1;
  int x_2 = 0;
  
  while (u != 1) {
    int q = v/u;
    int r = v - q*u;
    int x = x_2 - q*x_1;

    v = u;
    u = r;
    
    x_2 = x_1;
    x_1 = x;
  }

  return positive_modulo(x_1, p);
}

__global__ void mod_p_gaussian_backward_reduction(GaussianEliminationCtx *__restrict__ ctx, int *__restrict__ A, int n_rows, int n_cols, int curr_col, int *__restrict__ pivot_locations) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
 
  int pivot_row = pivot_locations[curr_col];
  
  if (curr_col + x + 1 >= n_cols || y >= pivot_row || pivot_row == -1) { //
    return;
  }
  
  int curr_entry = A[y * n_cols + curr_col + x + 1];
  curr_entry -= A[y * n_cols + curr_col] * A[pivot_row * n_cols + curr_col + x + 1] * 1;
  A[y * n_cols + curr_col + x + 1] = positive_modulo(curr_entry, (*ctx).prime_number); // TODO: performance, int size
}

__global__ void mod_p_gaussian_backward_clean_column(GaussianEliminationCtx *__restrict__ ctx, int *__restrict__ A, int n_rows, int n_cols, int curr_col, int *__restrict__ pivot_locations) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;

  int pivot_row = pivot_locations[curr_col];

  if (x >= pivot_row || pivot_row == -1 ) {
    return;
  }

  A[x*n_cols + curr_col] = 0;
}

__global__ void mod_p_gaussian_clean_column(GaussianEliminationCtx *__restrict__ ctx, int *__restrict__ A, int n_rows, int n_cols, int curr_col) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;

  if (x >= n_rows || x <= (*ctx).mod_p_pivot_seek_from_row-1 || (*ctx).mod_p_row_to_push == -1 ) {
    return;
  }

  A[x*n_cols + curr_col] = 0;
}

__global__ void mod_p_gaussian_elimination(GaussianEliminationCtx *__restrict__ ctx, int *__restrict__ A, int n_rows, int n_cols, int curr_col) { 
  /*
    Integer matrix routine.
  */
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int curr_row = blockIdx.y * blockDim.y + threadIdx.y;
  
  int mod_p_pivot_seek_from_row = (*ctx).mod_p_pivot_seek_from_row;

  // (*ctx).mod_p_row_to_push == -1: zero column
  if (curr_row <= mod_p_pivot_seek_from_row-1 || \
      curr_row >= n_rows || \
      curr_col + x + 1 >= n_cols || \
      (*ctx).mod_p_row_to_push == -1) {
    return;
  }

  int curr_entry = A[curr_row*n_cols + curr_col + x + 1];
  curr_entry -= (A[curr_row*n_cols + curr_col] * A[(mod_p_pivot_seek_from_row-1)*n_cols + curr_col + x + 1]) * 1;
  A[curr_row*n_cols + curr_col + x + 1] = positive_modulo(curr_entry, (*ctx).prime_number); // TODO: performance, int size
}

__global__ void mod_p_exchange_rows(GaussianEliminationCtx *__restrict__ ctx, int *__restrict__ A, int n_rows, int n_cols, int curr_col) { 
  /*
     Integer matrix routine.
  */
  int x = blockIdx.x * blockDim.x + threadIdx.x;

  int tmp_input;
  int src_row;
  int dst_row;

  src_row = (*ctx).mod_p_row_to_push;
  if (src_row == -1 || x + curr_col >= n_cols) {
      return;
  }

  dst_row = (*ctx).mod_p_pivot_seek_from_row - 1; // skip last found

  // mod_p_inverse for reduction purposes
  A[n_cols*src_row + curr_col + x] *= mod_p_inverse((*ctx).prime_number, (*ctx).mod_p_pivot_val);

  if (src_row == dst_row) {
      return;
  }

  tmp_input = A[n_cols*dst_row + curr_col + x];

  A[n_cols*dst_row + curr_col + x] = A[n_cols*src_row + curr_col + x]; 
  A[n_cols*src_row + curr_col + x] = tmp_input;
}

__global__ void mod_p_seek_row_to_push(GaussianEliminationCtx *__restrict__ ctx, int *__restrict__ A, int n_rows, int n_cols, int curr_col, int *__restrict__ pivot_locations) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;

  int pivot_candidate_val;

  if (x < (*ctx).mod_p_pivot_seek_from_row || x >= n_rows) {
      return;
  }

  if (atomicCAS(&((*ctx).mod_p_curr_col), curr_col - 1, curr_col) == curr_col - 1) { // TODO: check performance        
    (*ctx).mod_p_row_to_push = -1;                                                                                                                                    
    pivot_locations[curr_col] = -1;
  }

  pivot_candidate_val = A[x * n_cols + curr_col];

  if (pivot_candidate_val % (*ctx).prime_number != 0) { // TODO: slow for variable prime_number
    if (atomicCAS(&((*ctx).mod_p_row_to_push), -1, x) == -1) {
      pivot_locations[curr_col] = (*ctx).mod_p_pivot_seek_from_row;
      (*ctx).mod_p_pivot_seek_from_row += 1;
      (*ctx).mod_p_pivot_val = pivot_candidate_val;
    }
  }
}
  
// Host utils 

__host__ void print_matrix(int prime_number, vector<int> &matrix, int n_rows, int n_cols) { 
  /*
    Integer matrix routine
  */
 
  printf("[+] %u x %u matrix\n", n_rows, n_cols);
  
  for (int i = 0; i < n_rows; i++) {
    for (int j = 0; j < n_cols; j++) {
      printf("%d\t\t", ((matrix[i*n_cols+j] % prime_number) + prime_number) % prime_number);
    }
    printf("\n");
  }
  printf("\n");
}

// Main 

int main(int argc, char *argv[]) {
  // Initialize data: sample matrix 

  // size_t M_rows = 5; // 5x6
  // size_t M_cols = 6;

  size_t M_rows = 40000; // 5x6
  size_t M_cols = 80000;

  int prime_number = 5;

  vector<int> h_M(M_rows * M_cols, 1);

  generate(h_M.begin(), h_M.end(), [] {
    static int i = 0;
    int r = 0;

    int row = i / 40000;
    int col = i % 80000;

    if (i % 7 < 5 || i % 29 > 20) 
      r = (2 + i) % 5;
    if (col <= 10 || row > 40000 || row < 1)
      r = 0;
    i++;
    return r;
  });
  
  // h_M = {
  //    0, 0, 2, 3, 4, 3,
  //    0, 2, 3, 4, 0, 3,
  //    0, 3, 4, 0, 1, 3,
  //    0, 4, 0, 1, 2, 3,
  //    0, 0, 1, 2, 3, 3,
  // };

  print_matrix(prime_number, h_M, M_rows, M_cols);

  // Device TODO: split into separate procedures

  int h_M_size = M_rows*M_cols*sizeof(int);  
  int *d_M, *d_pivot_locations;
  GaussianEliminationCtx *d_ctx;

  hipMalloc(&d_M, h_M_size);
  hipMalloc(&d_pivot_locations, sizeof(int)*M_cols);
  hipMalloc(&d_ctx, sizeof(GaussianEliminationCtx));

  GaussianEliminationCtx h_ctx;
  h_ctx.prime_number = prime_number;
  h_ctx.mod_p_pivot_seek_from_row = 0;
  h_ctx.mod_p_curr_col = 0;
  h_ctx.mod_p_row_to_push = -1;

  hipMemcpy(d_M, h_M.data(), h_M_size, hipMemcpyHostToDevice);
  hipMemcpy(d_ctx, &h_ctx, sizeof(GaussianEliminationCtx), hipMemcpyHostToDevice);

  // Run kernels 
  
  dim3 num_threads_2d(DEFAULT_N_THREADS_PER_DIM, DEFAULT_N_THREADS_PER_DIM);
  int num_blocks;
  
  for (int j = 0; j < M_cols; j++) {
    dim3 num_blocks_2d((M_cols - j - 1) / DEFAULT_N_THREADS_PER_DIM + 1, M_rows / DEFAULT_N_THREADS_PER_DIM + 1);

    num_blocks = M_rows / DEFAULT_N_THREADS_PER_DIM + 1;
    mod_p_seek_row_to_push <<< num_blocks, DEFAULT_N_THREADS_PER_DIM >>> (d_ctx, d_M, M_rows, M_cols, j, d_pivot_locations);
    hipDeviceSynchronize();                         

    num_blocks = M_cols / DEFAULT_N_THREADS_PER_DIM + 1;               
    mod_p_exchange_rows <<< num_blocks, DEFAULT_N_THREADS_PER_DIM >>> (d_ctx, d_M, M_rows, M_cols, j);
    hipDeviceSynchronize();
        
    mod_p_gaussian_elimination <<< num_blocks_2d, num_threads_2d >>> (d_ctx, d_M, M_rows, M_cols, j); 
    hipDeviceSynchronize();
    
    num_blocks = M_rows / DEFAULT_N_THREADS_PER_DIM + 1;
    mod_p_gaussian_clean_column <<< num_blocks, DEFAULT_N_THREADS_PER_DIM >>> (d_ctx, d_M, M_rows, M_cols, j);
    hipDeviceSynchronize();
  }

  for (int i = 0; i < M_cols; i++) {
    dim3 num_blocks_2d((M_cols - i - 1) / DEFAULT_N_THREADS_PER_DIM + 1, M_rows / DEFAULT_N_THREADS_PER_DIM + 1);

    mod_p_gaussian_backward_reduction <<< num_blocks_2d, num_threads_2d >>> (d_ctx, d_M, M_rows, M_cols, i, d_pivot_locations);
    hipDeviceSynchronize();

    num_blocks = M_rows / DEFAULT_N_THREADS_PER_DIM + 1;
    mod_p_gaussian_backward_clean_column <<< num_blocks, DEFAULT_N_THREADS_PER_DIM >>> (d_ctx, d_M, M_rows, M_cols, i, d_pivot_locations); // to avoid more flow control
    hipDeviceSynchronize();
  }
  
  // Parse data 

  hipMemcpy(h_M.data(), d_M, h_M_size, hipMemcpyDeviceToHost);
  
  hipFree(d_M);
  hipFree(d_pivot_locations);
  hipFree(d_ctx);

  print_matrix(prime_number, h_M, M_rows, M_cols);

  // free(h_M);

  return 0;
}

