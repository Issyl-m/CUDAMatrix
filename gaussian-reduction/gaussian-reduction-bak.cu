/*
Copyright (c) 2025 Andrés Morán (andres.moran.l@uc.cl)
Licensed under the terms of the MIT License (see ./LICENSE).
*/

#include <hip/hip_runtime.h>

#include <cstdlib>
#include <iostream>
#include <vector>

using std::cout;
using std::vector;

// Constants

const int DEFAULT_N_THREADS_PER_DIM = 32; // max 1024 per block
const int DEFAULT_SHARED_MEM = 32*32 * 4 * 2; // 32768 bytes

// Structures

struct __align__(16) GaussianEliminationCtx {
  int prime_number;
  int mod_p_pivot_seek_from_row;
  int mod_p_curr_col;
  int mod_p_row_to_push;
  int mod_p_pivot_val; // TODO: possibly unused
};

// Kernels and devices 

/*
  Mod p linear algebra routines for small primes (<MAX_INT)
  TODO:
  - Separate routines for p >> 1, small primes and p = 2
  - Sparse matrix Gaussian elimination, multiplication
*/

__device__ int positive_modulo(int i, int n) { 
  /*
    Input: i arbitrary, n: modulus, n > 0
    Output: positive i % n representative
  */
  if (n == 2) {
    return i & 0x00000001;
  }
  return (i % n + n) % n;
}

__device__ int mod_2_inverse(int a) {
  return ((a & 0x00000001) << 1) - 1; // Output: -1 if error
}

__device__ int mod_3_inverse(int a) {
  int b = a % 3;
  if (b == 0) return -1;
  return b;
}

__device__ int mod_p_inverse(int p, int a) {
  /*
    Extended Euclidean division
    Mod p multiplicative inverse
    Output: x_1 = a^{-1}
  */
  int u = a;
  int v = p;
  
  if (p == 2)
    return mod_2_inverse(a);

  if (p == 3)
    return mod_3_inverse(a);

  if (u % v == 0) {
    return -1;
  }

  u = positive_modulo(a, p);
  
  int x_1 = 1;
  int x_2 = 0;
  
  while (u != 1) {
    int q = v/u;
    int r = v - q*u;
    int x = x_2 - q*x_1;

    v = u;
    u = r;
    
    x_2 = x_1;
    x_1 = x;
  }

  return positive_modulo(x_1, p);
}

__global__ void mod_p_gaussian_backward_substitution(GaussianEliminationCtx *__restrict__ ctx, int *__restrict__ A, int n_rows, int n_cols) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  
  if (x >= n_cols + 1) {
    return;
  }

  bool zeroing = true;

  int curr_row = -1;
  int prime_number = (*ctx).prime_number;

  for (int i = n_rows - 1; i >= 0; i--) {
    if (A[i * (n_cols + 1) + x] % prime_number != 0) { 
      for (int j = 0; j < n_cols + 1; j++) {
        if (j == x)
          continue;
        zeroing &= (A[i * (n_cols + 1) + j + x] % prime_number == 0);
      }
      if (zeroing) 
        curr_row = i;
      else
        return;
      break;
    }
  }

  if (zeroing and curr_row >= 0) {
    for (int i = curr_row + 1; i >= 0; i--) {
      A[i * (n_cols + 1) + x] = 0;
    }
  }
}

__global__ void mod_p_gaussian_clean_column(GaussianEliminationCtx *__restrict__ ctx, int *__restrict__ A, int n_rows, int n_cols, int curr_col) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;

  if (x >= n_rows || x <= (*ctx).mod_p_pivot_seek_from_row-1 || (*ctx).mod_p_row_to_push == -1 ) {
    return;
  }

  A[x*(n_cols+1) + curr_col] = 0;
}

__global__ void mod_p_gaussian_elimination(GaussianEliminationCtx *__restrict__ ctx, int *__restrict__ A, int n_rows, int n_cols, int curr_col) { 
  /*
    Integer matrix routine.
  */
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int curr_row = blockIdx.y * blockDim.y + threadIdx.y;
  
  // (*ctx).mod_p_row_to_push == -1: zero column
  if (curr_row <= (*ctx).mod_p_pivot_seek_from_row-1 || \
      curr_row >= n_rows || \
      curr_col + x + 1 >= n_cols || \
      (*ctx).mod_p_row_to_push == -1) {
    return;
  }
 
  A[curr_row*n_cols + curr_col + x + 1] -=\
    (A[curr_row*n_cols + curr_col] * A[((*ctx).mod_p_pivot_seek_from_row-1)*n_cols + curr_col + x + 1]);
  
  // A[curr_row*n_cols + curr_col + x + 1] = positive_modulo(A[curr_row*n_cols + curr_col + x + 1], (*ctx).prime_number);
}

__global__ void mod_p_exchange_rows(GaussianEliminationCtx *__restrict__ ctx, int *__restrict__ A, int n_rows, int n_cols, int curr_col) { 
  /*
     Integer matrix routine.
  */
  int x = blockIdx.x * blockDim.x + threadIdx.x;

  int tmp_input;
  int src_row;
  int dst_row;

  src_row = (*ctx).mod_p_row_to_push;
  if (src_row == -1 || x + curr_col >= n_cols) {
      return;
  }

  dst_row = (*ctx).mod_p_pivot_seek_from_row - 1; // skip last found

  // mod_p_inverse for reduction purposes
  A[n_cols*src_row + curr_col + x] *= mod_p_inverse((*ctx).prime_number, (*ctx).mod_p_pivot_val);

  if (src_row == dst_row) {
      return;
  }

  tmp_input = A[n_cols*dst_row + curr_col + x];

  A[n_cols*dst_row + curr_col + x] = A[n_cols*src_row + curr_col + x]; 
  A[n_cols*src_row + curr_col + x] = tmp_input;
}

__global__ void mod_p_seek_row_to_push(GaussianEliminationCtx *__restrict__ ctx, int *__restrict__ A, int n_rows, int n_cols, int curr_col) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;

  int pivot_candidate_val;

  if (x < (*ctx).mod_p_pivot_seek_from_row || x >= n_rows) {
      return;
  }

  if (atomicCAS(&((*ctx).mod_p_curr_col), curr_col - 1, curr_col) == curr_col - 1) { // TODO: check performance        
    (*ctx).mod_p_row_to_push = -1;                                                                                                                                    
  }

  pivot_candidate_val = A[x * n_cols + curr_col];

  if (pivot_candidate_val % (*ctx).prime_number != 0) { // TODO: slow for variable prime_number
    if (atomicCAS(&((*ctx).mod_p_row_to_push), -1, x) == -1) {
      (*ctx).mod_p_pivot_seek_from_row += 1;
      (*ctx).mod_p_pivot_val = pivot_candidate_val;
    }
  }
}
  
// Host utils 

__host__ void print_matrix(int prime_number, vector<int> &matrix, int n_rows, int n_cols) { 
  /*
    Integer matrix routine
  */
 
  printf("[+] %u x %u matrix\n", n_rows, n_cols);
  
  for (int i = 0; i < n_rows; i++) {
    for (int j = 0; j < n_cols; j++) {
      printf("%d\t\t", matrix[i*n_cols+j] % prime_number);
    }
    printf("\n");
  }
  printf("\n");
}

// Main 

int main(int argc, char *argv[]) {
  // Initialize data: sample matrix 

  size_t M_rows = 5;
  size_t M_cols = 6;

  int prime_number = 5;

  vector<int> h_M(M_rows * M_cols);
  
  h_M = {
    1, 2, 0, 4, 5, 6,
    0, 1, 0, 10, 11, 12,
    0, 0, 3, 0, 0, 0,
    0, 0, 0, 4, 5, 6,
    0, 0, 0, 0, 0, 6
  };

  print_matrix(prime_number, h_M, M_rows, M_cols);

  // Device TODO: split into separate procedures

  int h_M_size = M_rows*M_cols*sizeof(int);  
  int *d_M;
  GaussianEliminationCtx *d_ctx;

  hipMalloc(&d_M, h_M_size);
  hipMalloc(&d_ctx, sizeof(GaussianEliminationCtx));

  GaussianEliminationCtx h_ctx;
  h_ctx.prime_number = prime_number;
  h_ctx.mod_p_pivot_seek_from_row = 0;
  h_ctx.mod_p_curr_col = 0;
  h_ctx.mod_p_row_to_push = -1;

  hipMemcpy(d_M, h_M.data(), h_M_size, hipMemcpyHostToDevice);
  hipMemcpy(d_ctx, &h_ctx, sizeof(GaussianEliminationCtx), hipMemcpyHostToDevice);

  // Run kernels 
  
  dim3 num_threads_2d(DEFAULT_N_THREADS_PER_DIM, DEFAULT_N_THREADS_PER_DIM);
  int num_blocks;
  
  for (int j = 0; j < M_cols; j++) {
    dim3 num_blocks_2d((M_cols - j - 1) / DEFAULT_N_THREADS_PER_DIM + 1, M_rows / DEFAULT_N_THREADS_PER_DIM + 1);

    num_blocks = M_rows / DEFAULT_N_THREADS_PER_DIM + 1;
    mod_p_seek_row_to_push <<< num_blocks, DEFAULT_N_THREADS_PER_DIM >>> (d_ctx, d_M, M_rows, M_cols, j);
    hipDeviceSynchronize();                         

    num_blocks = M_cols / DEFAULT_N_THREADS_PER_DIM + 1;               
    mod_p_exchange_rows <<< num_blocks, DEFAULT_N_THREADS_PER_DIM >>> (d_ctx, d_M, M_rows, M_cols, j);
    hipDeviceSynchronize();
        
    mod_p_gaussian_elimination <<< num_blocks_2d, num_threads_2d >>> (d_ctx, d_M, M_rows, M_cols, j); 
    hipDeviceSynchronize();
    
    num_blocks = M_rows / DEFAULT_N_THREADS_PER_DIM + 1;
    mod_p_gaussian_clean_column <<< num_blocks, DEFAULT_N_THREADS_PER_DIM >>> (d_ctx, d_M, M_rows, M_rows, j);
    hipDeviceSynchronize();
  }

  num_blocks = M_cols / DEFAULT_N_THREADS_PER_DIM + 1;
  mod_p_gaussian_backward_substitution <<< num_blocks, DEFAULT_N_THREADS_PER_DIM >>> (d_ctx, d_M, M_rows, M_rows);
  hipDeviceSynchronize();
  
  // Parse data 

  hipMemcpy(h_M.data(), d_M, h_M_size, hipMemcpyDeviceToHost);
  
  hipFree(d_M);
  hipFree(d_ctx);

  print_matrix(prime_number, h_M, M_rows, M_cols);

  // free(h_M);

  return 0;
}

